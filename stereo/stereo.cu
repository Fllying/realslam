#include "hip/hip_runtime.h"
#include "stereo.h"

Stereo::Stereo() {
	this->BlockWidth = 32;
	this->BlockHeight = 12;
	this->StrideAlignment = 32;
}

Stereo::Stereo(int BlockWidth, int BlockHeight, int StrideAlignment) {
	this->BlockWidth = BlockWidth;
	this->BlockHeight = BlockHeight;
	this->StrideAlignment = StrideAlignment;
}

int Stereo::initializeOpticalFlow(int width, int height, int channels, int inputType, int nLevels, float scale, float lambda,
	float theta, float tau, int nWarpIters, int nSolverIters)
{
	//allocate all memories
	this->width = width;
	this->height = height;
	this->stride = iAlignUp(width);
	this->inputType = inputType;

	this->fScale = scale;
	this->nLevels = nLevels;
	this->inputChannels = channels;
	this->nSolverIters = nSolverIters; //number of inner iteration (ROF loop)
	this->nWarpIters = nWarpIters;

	this->lambda = lambda;
	this->theta = theta;
	this->tau = tau;

	pI0 = std::vector<float*>(nLevels);
	pI1 = std::vector<float*>(nLevels);
	pW = std::vector<int>(nLevels);
	pH = std::vector<int>(nLevels);
	pS = std::vector<int>(nLevels);
	pDataSize = std::vector<int>(nLevels);

	int newHeight = height;
	int newWidth = width;
	int newStride = iAlignUp(width);
	//std::cout << "Pyramid Sizes: " << newWidth << " " << newHeight << " " << newStride << std::endl;
	for (int level = 0; level < nLevels; level++) {
		pDataSize[level] = newStride * newHeight * sizeof(float);
		checkCudaErrors(hipMalloc(&pI0[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pI1[level], pDataSize[level]));
		
		pW[level] = newWidth;
		pH[level] = newHeight;
		pS[level] = newStride;
		newHeight = newHeight / fScale;
		newWidth = newWidth / fScale;
		newStride = iAlignUp(newWidth);
	}
	
	//runtime
	dataSize = stride * height * sizeof(float);
	dataSize8uc3 = stride * height * sizeof(uchar3);
	dataSize8u = stride * height * sizeof(uchar);
	dataSize32f = dataSize;
	dataSize32fc3 = dataSize * 3;
	checkCudaErrors(hipMalloc(&d_i1warp, dataSize));

	checkCudaErrors(hipMalloc(&d_du, dataSize));
	checkCudaErrors(hipMalloc(&d_dv, dataSize));
	checkCudaErrors(hipMalloc(&d_dus, dataSize));
	checkCudaErrors(hipMalloc(&d_dvs, dataSize));

	checkCudaErrors(hipMalloc(&d_dumed, dataSize));
	checkCudaErrors(hipMalloc(&d_dvmed, dataSize));
	checkCudaErrors(hipMalloc(&d_dumeds, dataSize));
	checkCudaErrors(hipMalloc(&d_dvmeds, dataSize));

	//dual TV
	checkCudaErrors(hipMalloc(&d_pu1, dataSize));
	checkCudaErrors(hipMalloc(&d_pu2, dataSize));
	checkCudaErrors(hipMalloc(&d_pv1, dataSize));
	checkCudaErrors(hipMalloc(&d_pv2, dataSize));
	//dual TV temps
	checkCudaErrors(hipMalloc(&d_pu1s, dataSize));
	checkCudaErrors(hipMalloc(&d_pu2s, dataSize));
	checkCudaErrors(hipMalloc(&d_pv1s, dataSize));
	checkCudaErrors(hipMalloc(&d_pv2s, dataSize));

	checkCudaErrors(hipMalloc(&d_Ix, dataSize));
	checkCudaErrors(hipMalloc(&d_Iy, dataSize));
	checkCudaErrors(hipMalloc(&d_Iz, dataSize));

	checkCudaErrors(hipMalloc(&d_u, dataSize));
	checkCudaErrors(hipMalloc(&d_v, dataSize));
	checkCudaErrors(hipMalloc(&d_us, dataSize));
	checkCudaErrors(hipMalloc(&d_vs, dataSize));

	if (inputType == CV_8UC3) {
		checkCudaErrors(hipMalloc(&d_i08uc3, dataSize8uc3));
		checkCudaErrors(hipMalloc(&d_i18uc3, dataSize8uc3));
	}
	else if (inputType == CV_8U) {
		checkCudaErrors(hipMalloc(&d_i08u, dataSize8u));
		checkCudaErrors(hipMalloc(&d_i18u, dataSize8u));
	}

	// colored uv, for display only
	checkCudaErrors(hipMalloc(&d_uvrgb, dataSize * 3));

	// Output mats
	uvrgb = cv::Mat(height, stride, CV_32FC3);
	upad = cv::Mat(height, stride, CV_32F);
	vpad = cv::Mat(height, stride, CV_32F);

	return 0;
}

int Stereo::initializeFisheyeStereo(int width, int height, int channels, int inputType, int nLevels, float scale, float lambda,
	float theta, float tau, int nWarpIters, int nSolverIters) {
	//allocate all memories
	this->width = width;
	this->height = height;
	this->stride = iAlignUp(width);
	this->inputType = inputType;

	this->fScale = scale;
	this->nLevels = nLevels;
	this->inputChannels = channels;
	this->nSolverIters = nSolverIters; //number of inner iteration (ROF loop)
	this->nWarpIters = nWarpIters;

	this->lambda = lambda;
	this->theta = theta;
	this->tau = tau;

	pI0 = std::vector<float*>(nLevels);
	pI1 = std::vector<float*>(nLevels);
	pW = std::vector<int>(nLevels);
	pH = std::vector<int>(nLevels);
	pS = std::vector<int>(nLevels);
	pDataSize = std::vector<int>(nLevels);
	pTvxForward = std::vector<float*>(nLevels);
	pTvyForward = std::vector<float*>(nLevels);
	pTvxBackward = std::vector<float*>(nLevels);
	pTvyBackward = std::vector<float*>(nLevels);

	int newHeight = height;
	int newWidth = width;
	int newStride = iAlignUp(width);
	//std::cout << "Pyramid Sizes: " << newWidth << " " << newHeight << " " << newStride << std::endl;
	for (int level = 0; level < nLevels; level++) {
		pDataSize[level] = newStride * newHeight * sizeof(float);
		checkCudaErrors(hipMalloc(&pI0[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pI1[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvxForward[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvyForward[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvxBackward[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvyBackward[level], pDataSize[level]));

		pW[level] = newWidth;
		pH[level] = newHeight;
		pS[level] = newStride;
		newHeight = newHeight / fScale;
		newWidth = newWidth / fScale;
		newStride = iAlignUp(newWidth);
	}

	//runtime
	dataSize = stride * height * sizeof(float);
	dataSize8uc3 = stride * height * sizeof(uchar3);
	dataSize8u = stride * height * sizeof(uchar);
	dataSize32f = dataSize;
	dataSize32fc3 = dataSize * 3;
	checkCudaErrors(hipMalloc(&d_i1warp, dataSize));

	checkCudaErrors(hipMalloc(&d_tvxForward, dataSize));
	checkCudaErrors(hipMalloc(&d_tvyForward, dataSize));
	checkCudaErrors(hipMalloc(&d_tvxBackward, dataSize));
	checkCudaErrors(hipMalloc(&d_tvyBackward, dataSize));
	checkCudaErrors(hipMalloc(&d_tvx2, dataSize));
	checkCudaErrors(hipMalloc(&d_tvy2, dataSize));
	checkCudaErrors(hipMalloc(&d_cvx, dataSize));
	checkCudaErrors(hipMalloc(&d_cvy, dataSize));

	checkCudaErrors(hipMalloc(&d_i1calibrated, dataSize));
	checkCudaErrors(hipMalloc(&d_Iw, dataSize));
	checkCudaErrors(hipMalloc(&d_Iz, dataSize));

	checkCudaErrors(hipMalloc(&d_w, dataSize));
	checkCudaErrors(hipMalloc(&d_wForward, dataSize));
	checkCudaErrors(hipMalloc(&d_wBackward, dataSize));
	checkCudaErrors(hipMalloc(&d_wFinal, dataSize));
	checkCudaErrors(hipMalloc(&d_u, dataSize));
	checkCudaErrors(hipMalloc(&d_v, dataSize));
	checkCudaErrors(hipMalloc(&d_uForward, dataSize));
	checkCudaErrors(hipMalloc(&d_vForward, dataSize));
	checkCudaErrors(hipMalloc(&d_us, dataSize));
	checkCudaErrors(hipMalloc(&d_vs, dataSize));
	checkCudaErrors(hipMalloc(&d_ws, dataSize));

	checkCudaErrors(hipMalloc(&d_du, dataSize));
	checkCudaErrors(hipMalloc(&d_dv, dataSize));
	checkCudaErrors(hipMalloc(&d_dw, dataSize));
	checkCudaErrors(hipMalloc(&d_dws, dataSize));
	checkCudaErrors(hipMalloc(&d_depth, dataSize));
	checkCudaErrors(hipMalloc(&d_depthFinal, dataSize));
	checkCudaErrors(hipMalloc(&d_occlusion, dataSize));

	checkCudaErrors(hipMalloc(&d_dwmed, dataSize));
	checkCudaErrors(hipMalloc(&d_dwmeds, dataSize));
	checkCudaErrors(hipMalloc(&d_pw1, dataSize));
	checkCudaErrors(hipMalloc(&d_pw2, dataSize));
	checkCudaErrors(hipMalloc(&d_pw1s, dataSize));
	checkCudaErrors(hipMalloc(&d_pw2s, dataSize));

	if (inputType == CV_8UC3) {
		checkCudaErrors(hipMalloc(&d_i08uc3, dataSize8uc3));
		checkCudaErrors(hipMalloc(&d_i18uc3, dataSize8uc3));
	}
	else if (inputType == CV_8U) {
		checkCudaErrors(hipMalloc(&d_i08u, dataSize8u));
		checkCudaErrors(hipMalloc(&d_i18u, dataSize8u));
	}

	// Plane sweep
	checkCudaErrors(hipMalloc(&ps_i1warp, dataSize));
	checkCudaErrors(hipMalloc(&ps_i1warps, dataSize));
	checkCudaErrors(hipMalloc(&ps_error, dataSize));
	checkCudaErrors(hipMalloc(&ps_depth, dataSize));
	checkCudaErrors(hipMalloc(&ps_disparity, dataSize));
	checkCudaErrors(hipMalloc(&ps_disparityForward, dataSize));
	checkCudaErrors(hipMalloc(&ps_disparityBackward, dataSize));
	checkCudaErrors(hipMalloc(&ps_disparityFinal, dataSize));

	// Colored uv, for display only
	checkCudaErrors(hipMalloc(&d_uvrgb, dataSize * 3));
	uvrgb = cv::Mat(height, stride, CV_32FC3);
	disparity = cv::Mat(height, stride, CV_32F);
	depth = cv::Mat(height, stride, CV_32F);
	planeSweepDepth = cv::Mat(height, stride, CV_32F);
	return 0;
}

int Stereo::loadVectorFields(cv::Mat translationVector, cv::Mat calibrationVector) {
	// Padding
	cv::Mat translationVectorPad = cv::Mat(height, stride, CV_32F);
	cv::Mat calibrationVectorPad = cv::Mat(height, stride, CV_32F);
	cv::copyMakeBorder(translationVector, translationVectorPad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);
	cv::copyMakeBorder(calibrationVector, calibrationVectorPad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);

	// Translation Vector Field
	translationVectorX = cv::Mat(height, stride, CV_32F);
	translationVectorY = cv::Mat(height, stride, CV_32F);
	calibrationVectorX = cv::Mat(height, stride, CV_32F);
	calibrationVectorY = cv::Mat(height, stride, CV_32F);

	cv::Mat tuv[2];
	cv::split(translationVectorPad, tuv);
	translationVectorX = tuv[0];
	translationVectorY = tuv[1];
	checkCudaErrors(hipMemcpy(d_tvxForward, (float *)translationVectorX.ptr(), dataSize32f, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tvyForward, (float *)translationVectorY.ptr(), dataSize32f, hipMemcpyHostToDevice));

	pTvxForward[0] = d_tvxForward;
	pTvyForward[0] = d_tvyForward;
	ScalarMultiply(d_tvxForward, -1.0f, width, height, stride, d_tvxBackward);
	ScalarMultiply(d_tvyForward, -1.0f, width, height, stride, d_tvyBackward);
	pTvxBackward[0] = d_tvxBackward;
	pTvyBackward[0] = d_tvyBackward;
	for (int level = 1; level < nLevels; level++) {
		//std::cout << pW[level] << " " << pH[level] << " " << pS[level] << std::endl;
		Downscale(pTvxForward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvxForward[level]);
		Downscale(pTvyForward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvyForward[level]);

		Downscale(pTvxBackward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvxBackward[level]);
		Downscale(pTvyBackward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvyBackward[level]);

	}

	// Calibration Vector Field
	cv::Mat cuv[2];
	cv::split(calibrationVectorPad, cuv);
	calibrationVectorX = cuv[0].clone();
	calibrationVectorY = cuv[1].clone();
	checkCudaErrors(hipMemcpy(d_cvx, (float *)calibrationVectorX.ptr(), dataSize32f, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cvy, (float *)calibrationVectorY.ptr(), dataSize32f, hipMemcpyHostToDevice));
	return 0;
}

int Stereo::copyImagesToDevice(cv::Mat i0, cv::Mat i1) {
	// Padding
	cv::copyMakeBorder(i0, im0pad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);
	cv::copyMakeBorder(i1, im1pad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);

	if (inputType == CV_8U) {
		checkCudaErrors(hipMemcpy(d_i08u, (uchar *)im0pad.ptr(), dataSize8u, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_i18u, (uchar *)im1pad.ptr(), dataSize8u, hipMemcpyHostToDevice));
		// Convert to 32F
		Cv8uToGray(d_i08u, pI0[0], width, height, stride);
		Cv8uToGray(d_i18u, pI1[0], width, height, stride);
	}
	else if (inputType == CV_32F) {
		checkCudaErrors(hipMemcpy(pI0[0], (float *)im0pad.ptr(), dataSize32f, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(pI1[0], (float *)im1pad.ptr(), dataSize32f, hipMemcpyHostToDevice));
	}
	else {
		checkCudaErrors(hipMemcpy(d_i08uc3, (uchar3 *)im0pad.ptr(), dataSize8uc3, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_i18uc3, (uchar3 *)im1pad.ptr(), dataSize8uc3, hipMemcpyHostToDevice));
		// Convert to 32F
		Cv8uc3ToGray(d_i08uc3, pI0[0], width, height, stride);
		Cv8uc3ToGray(d_i18uc3, pI1[0], width, height, stride);
	}
	return 0;
}

int Stereo::solveStereoForward() {
	// Warp i1 using vector fields
	WarpImage(pI1[0], width, height, stride, d_cvx, d_cvy, d_i1calibrated);
	Swap(pI1[0], d_i1calibrated);

	checkCudaErrors(hipMemset(d_w, 0, dataSize));
	checkCudaErrors(hipMemset(d_u, 0, dataSize));
	checkCudaErrors(hipMemset(d_v, 0, dataSize));
	// Construct pyramid
	for (int level = 1; level < nLevels; level++) {
		Downscale(pI0[level - 1],
			pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level],
			pI0[level]);

		Downscale(pI1[level - 1],
			pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level],
			pI1[level]);
	}

	//planeSweepForward();

	ComputeOpticalFlowVector(d_w, d_tvxForward, d_tvyForward, pW[0], pH[0], pS[0], d_u, d_v);

	/*cv::Mat calibrated = cv::Mat(height, stride, CV_32F);
	checkCudaErrors(hipMemcpy((float *)calibrated.ptr(), ps_disparity, width * height * sizeof(float), hipMemcpyDeviceToHost));
	cv::imshow("calibrated", calibrated/(float)planeSweepMaxDisparity);*/

	// Solve stereo
	for (int level = nLevels - 1; level >= 0; level--) {
		for (int warpIter = 0; warpIter < nWarpIters; warpIter++) {
			// Compute U,V from W d_w is magnitude of vector d_tvx, d_tvy
			// Warp using U,V
			//std::cout << "entered" << std::endl;
			checkCudaErrors(hipMemset(d_du, 0, dataSize));
			checkCudaErrors(hipMemset(d_dv, 0, dataSize));
			checkCudaErrors(hipMemset(d_dw, 0, dataSize));

			checkCudaErrors(hipMemset(d_dws, 0, dataSize));
			checkCudaErrors(hipMemset(d_dwmed, 0, dataSize));
			checkCudaErrors(hipMemset(d_dwmeds, 0, dataSize));
			checkCudaErrors(hipMemset(d_pw1, 0, dataSize));
			checkCudaErrors(hipMemset(d_pw2, 0, dataSize));

			FindWarpingVector(d_u, d_v, pTvxForward[level], pTvyForward[level], pW[level], pH[level], pS[level], d_tvx2, d_tvy2);
			WarpImage(pI1[level], pW[level], pH[level], pS[level], d_u, d_v, d_i1warp);
			//std::cout << pW[level] << " " << pH[level] << " " << pS[level] << std::endl;
			ComputeDerivativesFisheye(pI0[level], d_i1warp, pTvxForward[level], pTvyForward[level], pW[level], pH[level], pS[level], d_Iw, d_Iz);
			/*if (level == 0) {
				cv::Mat calibrated = cv::Mat(pH[level], pS[level], CV_32F);
				checkCudaErrors(hipMemcpy((float *)calibrated.ptr(), d_i1warp, pS[level] * pH[level] * sizeof(float), hipMemcpyDeviceToHost));
				cv::imshow("gradient", calibrated);
			}*/

			// Inner iteration
			for (int iter = 0; iter < nSolverIters; ++iter)
			{
				SolveDataL1Stereo(d_dwmed,
					d_pw1, d_pw2,
					d_Iw, d_Iz,
					pW[level], pH[level], pS[level],
					lambda, theta,
					d_dwmeds); //du1 = duhat output
				Swap(d_dwmed, d_dwmeds);

				SolveSmoothDualTVGlobalStereo(d_dwmed,
					d_pw1, d_pw2,
					pW[level], pH[level], pS[level],
					tau, theta,
					d_pw1s, d_pw2s);
				Swap(d_pw1, d_pw1s);
				Swap(d_pw2, d_pw2s);
			}

			// Sanity Check: Limit disparity to 1
			LimitRange(d_dwmed, 1.0f, pW[level], pH[level], pS[level], d_dwmeds);
			Swap(d_dwmed, d_dwmeds);

			//// One median filtering
			MedianFilterDisparity(d_dwmed, pW[level], pH[level], pS[level],
				d_dwmeds, 5);
			Swap(d_dwmed, d_dwmeds);

			//// Calculate d_du, d_dv
			ComputeOpticalFlowVector(d_dwmed, d_tvx2, d_tvy2, pW[level], pH[level], pS[level], d_du, d_dv);

			//// update w, u, v
			Add(d_w, d_dwmed, pH[level] * pS[level], d_w);
			Add(d_u, d_du, pH[level] * pS[level], d_u);
			Add(d_v, d_dv, pH[level] * pS[level], d_v);
		}

		// Upscale
		if (level > 0)
		{
			float scale = fScale;
			Upscale(d_u, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_us);
			Upscale(d_v, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_vs);
			Upscale(d_w, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_ws);
			Swap(d_u, d_us);
			Swap(d_v, d_vs);
			Swap(d_w, d_ws);
		}
	}

	Clone(d_w, width, height, stride, d_wForward);

	if (visualizeResults) {
		FlowToHSV(d_u, d_v, width, height, stride, d_uvrgb, flowScale);
	}

	return 0;
}

int Stereo::solveStereoBackward() {
	// Warp i1 using vector fields
	//WarpImage(pI1[0], width, height, stride, d_cvx, d_cvy, d_i1calibrated);
	//Swap(pI1[0], d_i1calibrated);
	Swap(pI0[0], pI1[0]);
	
	checkCudaErrors(hipMemset(d_w, 0, dataSize));
	checkCudaErrors(hipMemset(d_u, 0, dataSize));
	checkCudaErrors(hipMemset(d_v, 0, dataSize));
	// Construct pyramid
	for (int level = 1; level < nLevels; level++) {
		Swap(pI0[level], pI1[level]);
	}

	//planeSweepBackward();

	//Clone(ps_disparity, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], d_w);
	ComputeOpticalFlowVector(d_w, d_tvxBackward, d_tvyBackward, pW[0], pH[0], pS[0], d_u, d_v);

	/*cv::Mat calibrated = cv::Mat(height, stride, CV_32F);
	checkCudaErrors(hipMemcpy((float *)calibrated.ptr(), ps_disparity, width * height * sizeof(float), hipMemcpyDeviceToHost));
	cv::imshow("calibrated", calibrated/(float)planeSweepMaxDisparity);*/

	// Solve stereo
	for (int level = nLevels - 1; level >= 0; level--) {
		for (int warpIter = 0; warpIter < nWarpIters; warpIter++) {
			// Compute U,V from W d_w is magnitude of vector d_tvx, d_tvy
			// Warp using U,V
			//std::cout << "entered" << std::endl;
			checkCudaErrors(hipMemset(d_du, 0, dataSize));
			checkCudaErrors(hipMemset(d_dv, 0, dataSize));
			checkCudaErrors(hipMemset(d_dw, 0, dataSize));

			checkCudaErrors(hipMemset(d_dws, 0, dataSize));
			checkCudaErrors(hipMemset(d_dwmed, 0, dataSize));
			checkCudaErrors(hipMemset(d_dwmeds, 0, dataSize));
			checkCudaErrors(hipMemset(d_pw1, 0, dataSize));
			checkCudaErrors(hipMemset(d_pw2, 0, dataSize));

			FindWarpingVector(d_u, d_v, pTvxBackward[level], pTvyBackward[level], 
				pW[level], pH[level], pS[level], d_tvx2, d_tvy2);
			WarpImage(pI1[level], pW[level], pH[level], pS[level], d_u, d_v, d_i1warp);
			//std::cout << pW[level] << " " << pH[level] << " " << pS[level] << std::endl;
			ComputeDerivativesFisheye(pI0[level], d_i1warp, pTvxBackward[level], pTvyBackward[level], 
				pW[level], pH[level], pS[level], d_Iw, d_Iz);
			/*if (level == 0) {
				cv::Mat calibrated = cv::Mat(pH[level], pS[level], CV_32F);
				checkCudaErrors(hipMemcpy((float *)calibrated.ptr(), d_i1warp, pS[level] * pH[level] * sizeof(float), hipMemcpyDeviceToHost));
				cv::imshow("gradient", calibrated);
			}*/

			// Inner iteration
			for (int iter = 0; iter < nSolverIters; ++iter)
			{
				SolveDataL1Stereo(d_dwmed, 
					d_pw1, d_pw2,
					d_Iw, d_Iz,
					pW[level], pH[level], pS[level],
					lambda, theta,
					d_dwmeds); //du1 = duhat output
				Swap(d_dwmed, d_dwmeds);

				SolveSmoothDualTVGlobalStereo(d_dwmed, 
					d_pw1, d_pw2,
					pW[level], pH[level], pS[level],
					tau, theta,
					d_pw1s, d_pw2s);
				Swap(d_pw1, d_pw1s);
				Swap(d_pw2, d_pw2s);
			}

			// Sanity Check: Limit disparity to 1
			LimitRange(d_dwmed, 1.0f, pW[level], pH[level], pS[level], d_dwmeds);
			Swap(d_dwmed, d_dwmeds);

			//// One median filtering
			MedianFilterDisparity(d_dwmed, pW[level], pH[level], pS[level],
				d_dwmeds, 5);
			Swap(d_dwmed, d_dwmeds);

			//// Calculate d_du, d_dv
			ComputeOpticalFlowVector(d_dwmed, d_tvx2, d_tvy2, pW[level], pH[level], pS[level], d_du, d_dv);

			//// update w, u, v
			Add(d_w, d_dwmed, pH[level] * pS[level], d_w);
			Add(d_u, d_du, pH[level] * pS[level], d_u);
			Add(d_v, d_dv, pH[level] * pS[level], d_v);
		}

		// Upscale
		if (level > 0)
		{
			float scale = fScale;
			Upscale(d_u, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_us);
			Upscale(d_v, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_vs);
			Upscale(d_w, pW[level], pH[level], pS[level], pW[level - 1], pH[level - 1], pS[level - 1], scale, d_ws);
			Swap(d_u, d_us);
			Swap(d_v, d_vs);
			Swap(d_w, d_ws);
		}
	}

	Clone(d_w, width, height, stride, d_wBackward);

	if (visualizeResults) {
		FlowToHSV(d_u, d_v, width, height, stride, d_uvrgb, flowScale);
	}

	return 0;
}

int Stereo::occlusionCheck(float threshold) {
	isOcclusionChecked = true;
	// Get wFinal
	OcclusionCheck(d_wForward, d_wBackward, threshold, d_uForward, d_vForward, width, height, stride, d_wFinal);
	return 0;
}

int Stereo::planeSweepForward() {
	// Plane sweep on level=1
	int planeSweepLevel = 0;
	checkCudaErrors(hipMemset(ps_error, 0, dataSize));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize));
	checkCudaErrors(hipMemset(ps_disparity, 0, dataSize));
	Clone(pI1[planeSweepLevel], pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_i1warp);
	SetValue(ps_error, planeSweepMaxError, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI0[planeSweepLevel], ps_disparity, sweep, planeSweepWindow,
			pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], d_tvxForward, d_tvyForward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}
	//Clone(ps_disparity, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], d_w);
	return 0;
}

int Stereo::planeSweepBackward() {
	// Plane sweep on level=1
	int planeSweepLevel = 0;
	checkCudaErrors(hipMemset(ps_error, 0, dataSize));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize));
	checkCudaErrors(hipMemset(ps_disparity, 0, dataSize));
	Clone(pI1[planeSweepLevel], pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_i1warp);
	SetValue(ps_error, planeSweepMaxError, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI0[planeSweepLevel], ps_disparity, sweep, planeSweepWindow,
			pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel],
				d_tvxBackward, d_tvyBackward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}
	return 0;
}

int Stereo::planeSweepOcclusionCheck() {
	isPlaneSweepOcclusionChecked = true;
	// Get wFinal

	return 0;
}

int Stereo::copyStereoToHost(cv::Mat &wCropped) {
	// Convert Disparity to Depth
	if (isOcclusionChecked) {
		ConvertDisparityToDepth(d_wFinal, baseline, focal, width, height, stride, d_depth);
	}
	else {
		ConvertDisparityToDepth(d_w, baseline, focal, width, height, stride, d_depth);
	}
	
	// Remove Padding
	//checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_w, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_depth, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	wCropped = depth(roi);
	return 0;
}

int Stereo::copyPlaneSweepToHost(cv::Mat &ps) {
	// Convert Disparity to Depth
	if (isPlaneSweepOcclusionChecked) {
		ConvertDisparityToDepth(ps_disparityFinal, baseline, focal, width, height, stride, ps_depth);
	}
	else {
		ConvertDisparityToDepth(ps_disparity, baseline, focal, width, height, stride, ps_depth);
	}
	
	// Remove Padding
	//checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_w, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)planeSweepDepth.ptr(), ps_depth, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	ps = planeSweepDepth(roi);
	return 0;
}

int Stereo::solveOpticalFlow() {
	// construct pyramid
	for (int level = 1; level < nLevels; level++) {
		Downscale(pI0[level - 1],
			pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level],
			pI0[level]);

		Downscale(pI1[level - 1],
			pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level],
			pI1[level]);
	}

	// solve flow
	checkCudaErrors(hipMemset(d_u, 0, dataSize));
	checkCudaErrors(hipMemset(d_v, 0, dataSize));

	for (int level = nLevels - 1; level >= 0; level--) {
		for (int warpIter = 0; warpIter < nWarpIters; warpIter++) {
			//std::cout << level << std::endl;
			//initialize zeros
			checkCudaErrors(hipMemset(d_du, 0, dataSize));
			checkCudaErrors(hipMemset(d_dv, 0, dataSize));
			checkCudaErrors(hipMemset(d_dus, 0, dataSize));
			checkCudaErrors(hipMemset(d_dvs, 0, dataSize));

			checkCudaErrors(hipMemset(d_dumed, 0, dataSize));
			checkCudaErrors(hipMemset(d_dvmed, 0, dataSize));
			checkCudaErrors(hipMemset(d_dumeds, 0, dataSize));
			checkCudaErrors(hipMemset(d_dvmeds, 0, dataSize));

			checkCudaErrors(hipMemset(d_pu1, 0, dataSize));
			checkCudaErrors(hipMemset(d_pu2, 0, dataSize));
			checkCudaErrors(hipMemset(d_pv1, 0, dataSize));
			checkCudaErrors(hipMemset(d_pv2, 0, dataSize));

			//warp frame 1
			WarpImage(pI1[level], pW[level], pH[level], pS[level], d_u, d_v, d_i1warp);
			ComputeDerivatives(pI0[level], d_i1warp, pW[level], pH[level], pS[level], d_Ix, d_Iy, d_Iz);

			//inner iteration
			for (int iter = 0; iter < nSolverIters; ++iter)
			{
				SolveDataL1(d_dumed, d_dvmed,
					d_pu1, d_pu2,
					d_pv1, d_pv2,
					d_Ix, d_Iy, d_Iz,
					pW[level], pH[level], pS[level],
					lambda, theta,
					d_dumeds, d_dvmeds); //du1 = duhat output
				Swap(d_dumed, d_dumeds);
				Swap(d_dvmed, d_dvmeds);

				SolveSmoothDualTVGlobal(d_dumed, d_dvmed,
					d_pu1, d_pu2, d_pv1, d_pv2,
					pW[level], pH[level], pS[level],
					tau, theta,
					d_pu1s, d_pu2s, d_pv1s, d_pv2s);
				Swap(d_pu1, d_pu1s);
				Swap(d_pu2, d_pu2s);
				Swap(d_pv1, d_pv1s);
				Swap(d_pv2, d_pv2s);
				//***********************************

				/*MedianFilter(d_dumed, d_dvmed, pW[level], pH[level], pS[level],
					d_dumeds, d_dvmeds, 5);
				Swap(d_dumed, d_dumeds);
				Swap(d_dvmed, d_dvmeds);*/
			}
			// one median filtering
			MedianFilter(d_dumed, d_dvmed, pW[level], pH[level], pS[level],
				d_dumeds, d_dvmeds, 5);
			Swap(d_dumed, d_dumeds);
			Swap(d_dvmed, d_dvmeds);

			// update u, v
			Add(d_u, d_dumed, pH[level] * pS[level], d_u);
			Add(d_v, d_dvmed, pH[level] * pS[level], d_v);
			/*
						MedianFilter(d_u, d_v, pW[level], pH[level], pS[level],
							d_dumeds, d_dvmeds, 5);
						Swap(d_u, d_dumeds);
						Swap(d_v, d_dvmeds);*/
		}

		//upscale
		if (level > 0)
		{
			// scale uv
			//float scale = (float)pW[level + 1] / (float)pW[level];
			float scale = fScale;

			Upscale(d_u, pW[level], pH[level], pS[level],
				pW[level - 1], pH[level - 1], pS[level - 1], scale, d_us);

			//float scaleY = (float)pH[level + 1] / (float)pH[level];

			Upscale(d_v, pW[level], pH[level], pS[level],
				pW[level - 1], pH[level - 1], pS[level - 1], scale, d_vs);

			Swap(d_u, d_us);
			Swap(d_v, d_vs);
		}
	}
	
	if (visualizeResults) {
		FlowToHSV(d_u, d_v, width, height, stride, d_uvrgb, flowScale);
	}
	//FlowToHSV(d_u, d_v, width, height, stride, d_uvrgb, flowScale);
	//SolveSceneFlow(d_u, d_v, d_depth016u, d_depth116u, width, height, stride, d_sceneflow);
	//std::cout << stride << " " << height << " " << height << " " << inputChannels << std::endl;
	return 0;
}


int Stereo::copyOpticalFlowVisToHost(cv::Mat &uvrgbCropped) {
	// Remove Padding
	checkCudaErrors(hipMemcpy((float3 *)uvrgb.ptr(), d_uvrgb, width * height * sizeof(float) * 3, hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	uvrgbCropped = uvrgb(roi);
	return 0;
}

int Stereo::copyOpticalFlowToHost(cv::Mat &u, cv::Mat &v) {
	checkCudaErrors(hipMemcpy((float *)upad.ptr(), d_u, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)vpad.ptr(), d_v, stride * height * sizeof(float), hipMemcpyDeviceToHost));

	// Remove Padding
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	u = upad(roi);
	v = vpad(roi);

	return 0;
}

// Align up n to the nearest multiple of m
inline int Stereo::iAlignUp(int n)
{
	int m = this->StrideAlignment;
	int mod = n % m;

	if (mod)
		return n + m - mod;
	else
		return n;
}

int Stereo::iDivUp(int n, int m)
{
	return (n + m - 1) / m;
}

// swap two values
template<typename T>
inline void Stereo::Swap(T &a, T &ax)
{
	T t = a;
	a = ax;
	ax = t;
}

//swap four values
template<typename T>
inline void Stereo::Swap(T &a, T &ax, T &b, T &bx)
{
	Swap(a, ax);
	Swap(b, bx);
}

//swap eight values
template<typename T>
inline void Stereo::Swap(T &a, T &ax, T &b, T &bx, T &c, T &cx, T &d, T &dx)
{
	Swap(a, ax);
	Swap(b, bx);
	Swap(c, cx);
	Swap(d, dx);
}

int Stereo::computePyramidLevels(int width, int height, int minWidth, float scale) {
	int nLevels = 1;
	int pHeight = (int)((float)height / scale);
	while (pHeight > minWidth) {
		nLevels++;
		pHeight = (int)((float)pHeight / scale);
	}
	std::cout << "Pyramid Levels: " << nLevels << std::endl;
	return nLevels;
}

int Stereo::initializeColorWheel() {
	checkCudaErrors(hipMalloc(&d_colorwheel, 55 * 3 * sizeof(float)));
	float colorwheel[165] = { 255, 0, 0,
		255, 17, 0,
		255, 34, 0,
		255, 51, 0,
		255, 68, 0,
		255, 85, 0,
		255, 102, 0,
		255, 119, 0,
		255, 136, 0,
		255, 153, 0,
		255, 170, 0,
		255, 187, 0,
		255, 204, 0,
		255, 221, 0,
		255, 238, 0,
		255, 255, 0,
		213, 255, 0,
		170, 255, 0,
		128, 255, 0,
		85, 255, 0,
		43, 255, 0,
		0, 255, 0,
		0, 255, 63,
		0, 255, 127,
		0, 255, 191,
		0, 255, 255,
		0, 232, 255,
		0, 209, 255,
		0, 186, 255,
		0, 163, 255,
		0, 140, 255,
		0, 116, 255,
		0, 93, 255,
		0, 70, 255,
		0, 47, 255,
		0, 24, 255,
		0, 0, 255,
		19, 0, 255,
		39, 0, 255,
		58, 0, 255,
		78, 0, 255,
		98, 0, 255,
		117, 0, 255,
		137, 0, 255,
		156, 0, 255,
		176, 0, 255,
		196, 0, 255,
		215, 0, 255,
		235, 0, 255,
		255, 0, 255,
		255, 0, 213,
		255, 0, 170,
		255, 0, 128,
		255, 0, 85,
		255, 0, 43 };
	checkCudaErrors(hipMemcpy(colorwheel, d_colorwheel, 55 * 3 * sizeof(float), hipMemcpyDeviceToHost));
	return 0;
}