#include "hip/hip_runtime.h"
#include "stereoLite.h"

__global__ void LiteCloneKernel(float* src, float* dst, int width, int height, int stride) {
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;
		dst[pos] = src[pos];
	}
}

void StereoLite::Clone(float* src, float* dst, int w, int h, int s) {
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));
	LiteCloneKernel << < blocks, threads >> > (src, dst, w, h, s);
}


__global__ void LiteCloneKernel2(float2* src, float2* dst, int width, int height, int stride) {
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;
		dst[pos] = src[pos];
	}
}

void StereoLite::Clone(float2* src, float2* dst, int w, int h, int s) {
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));
	LiteCloneKernel2 << < blocks, threads >> > (src, dst, w, h, s);
}

// Set Value
__global__
void LiteSetValueKernel(float *image, float value, int width, int height, int stride)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	image[pos] = value;
}

void StereoLite::SetValue(float *image, float value, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));
	LiteSetValueKernel << < blocks, threads >> > (image, value, w, h, s);
}