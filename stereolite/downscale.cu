#include "hip/hip_runtime.h"
#include "stereoLite.h"

/// image to downscale
texture<float, hipTextureType2D, hipReadModeElementType> texFine;
texture<float2, hipTextureType2D, hipReadModeElementType> texFineFloat2;

// *********************************
// Downscaling
// *********************************
__global__ void LiteDownscaleKernel(int width, int height, int stride, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}

void StereoLite::Downscale(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModeLinear;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	LiteDownscaleKernel << < blocks, threads >> > (newWidth, newHeight, newStride, out);
}


__global__ void LiteDownscaleNearestNeighborKernel(int width, int height, int stride, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = tex2D(texFine, x, y);
	/*out[pos] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));*/
}

void StereoLite::DownscaleNearestNeighbor(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModePoint;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	LiteDownscaleNearestNeighborKernel << < blocks, threads >> > (newWidth, newHeight, newStride, out);
}


// *********************************
// Downscaling for Float2
// *********************************
__global__ void LiteDownscaleKernel(int width, int height, int stride, float2 *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	float2 val00 = tex2D(texFineFloat2, x - dx * 0.25f, y);
	float2 val01 = tex2D(texFineFloat2, x + dx * 0.25f, y);
	float2 val10 = tex2D(texFineFloat2, x, y - dy * 0.25f);
	float2 val11 = tex2D(texFineFloat2, x, y + dy * 0.25f);
	out[pos].x = 0.25f * (val00.x + val01.x + val10.x + val11.x);
	out[pos].y = 0.25f * (val00.y + val01.y + val10.y + val11.y);
}

void StereoLite::Downscale(const float2 *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float2 *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFineFloat2.addressMode[0] = hipAddressModeMirror;
	texFineFloat2.addressMode[1] = hipAddressModeMirror;
	texFineFloat2.filterMode = hipFilterModeLinear;
	texFineFloat2.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

	checkCudaErrors(hipBindTexture2D(0, texFineFloat2, src, width, height, stride * sizeof(float2)));

	LiteDownscaleKernel << < blocks, threads >> > (newWidth, newHeight, newStride, out);
}


// ***********************************
// Downscale with vector downscaling
//************************************

__global__ void LiteDownscaleScalingKernel(int width, int height, int stride, float scale, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = scale * 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}

void StereoLite::Downscale(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float scale, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModeLinear;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	LiteDownscaleScalingKernel << < blocks, threads >> > (newWidth, newHeight, newStride, scale, out);
}


// ***********************************
// Downscale with vector downscaling for Float2
//************************************

__global__ void LiteDownscaleScalingKernel(int width, int height, int stride, float scale, float2 *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	float2 val00 = tex2D(texFineFloat2, x - dx * 0.25f, y);
	float2 val01 = tex2D(texFineFloat2, x + dx * 0.25f, y);
	float2 val10 = tex2D(texFineFloat2, x, y - dy * 0.25f);
	float2 val11 = tex2D(texFineFloat2, x, y + dy * 0.25f);
	out[pos].x = scale * 0.25f * (val00.x + val01.x + val10.x + val11.x);
	out[pos].y = scale * 0.25f * (val00.y + val01.y + val10.y + val11.y);
}

void StereoLite::Downscale(const float2 *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float scale, float2 *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFineFloat2.addressMode[0] = hipAddressModeMirror;
	texFineFloat2.addressMode[1] = hipAddressModeMirror;
	texFineFloat2.filterMode = hipFilterModeLinear;
	texFineFloat2.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

	checkCudaErrors(hipBindTexture2D(0, texFineFloat2, src, width, height, stride * sizeof(float2)));

	LiteDownscaleScalingKernel << < blocks, threads >> > (newWidth, newHeight, newStride, scale, out);
}