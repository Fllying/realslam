#include "hip/hip_runtime.h"
#include "stereolite.h"

StereoLite::StereoLite() {
	this->BlockHeight = 12;
	this->BlockWidth = 32;
	this->StrideAlignment = 32;
}

StereoLite::StereoLite(int blockWidth, int blockHeight, int strideAlignment) {
	this->BlockHeight = blockHeight;
	this->BlockWidth = blockWidth;
	this->StrideAlignment = strideAlignment;
}

int StereoLite::initialize(int width, int height, float lambda, float theta, float tau,
	int nLevels, float fScale, int nWarpIters, int nSolverIters) {

	this->width = width;
	this->height = height;
	this->stride = this->iAlignUp(width);
	this->lambda = lambda;
	this->theta = theta;
	this->tau = tau;
	this->fScale = fScale;
	this->nLevels = nLevels;
	this->nWarpIters = nWarpIters;
	this->nSolverIters = nSolverIters;

	pI0 = std::vector<float*>(nLevels);
	pI1 = std::vector<float*>(nLevels);
	pW = std::vector<int>(nLevels);
	pH = std::vector<int>(nLevels);
	pS = std::vector<int>(nLevels);
	pDataSize = std::vector<int>(nLevels);
	pTvForward = std::vector<float2*>(nLevels);
	pTvBackward = std::vector<float2*>(nLevels);
	pFisheyeMask = std::vector<float*>(nLevels);

	int newHeight = height;
	int newWidth = width;
	int newStride = iAlignUp(width);
	//std::cout << "Pyramid Sizes: " << newWidth << " " << newHeight << " " << newStride << std::endl;
	for (int level = 0; level < nLevels; level++) {
		pDataSize[level] = newStride * newHeight * sizeof(float);
		checkCudaErrors(hipMalloc(&pI0[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pI1[level], pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvForward[level], 2 * pDataSize[level]));
		checkCudaErrors(hipMalloc(&pTvBackward[level], 2 * pDataSize[level]));
		checkCudaErrors(hipMalloc(&pFisheyeMask[level], pDataSize[level]));

		//std::cout << newHeight << " " << newWidth << " " << newStride << std::endl;

		pW[level] = newWidth;
		pH[level] = newHeight;
		pS[level] = newStride;
		newHeight = (int)((float)newHeight / fScale);
		newWidth = (int)((float)newWidth / fScale);
		newStride = iAlignUp(newWidth);
	}

	//std::cout << stride << " " << height << std::endl;
	dataSize8u = stride * height * sizeof(uchar);
	dataSize8uc3 = stride * height * sizeof(uchar3);
	dataSize32f = stride * height * sizeof(float);
	dataSize32fc2 = stride * height * sizeof(float2);
	dataSize32fc3 = stride * height * sizeof(float3);
	dataSize32fc4 = stride * height * sizeof(float4);

	// Inputs and Outputs
	checkCudaErrors(hipMalloc(&d_i0, dataSize32f));
	checkCudaErrors(hipMalloc(&d_i1, dataSize32f));
	checkCudaErrors(hipMalloc(&d_i1warp, dataSize32f));
	checkCudaErrors(hipMalloc(&d_i08u, dataSize8u));
	checkCudaErrors(hipMalloc(&d_i18u, dataSize8u));
	checkCudaErrors(hipMalloc(&d_i08uc3, dataSize8uc3));
	checkCudaErrors(hipMalloc(&d_i18uc3, dataSize8uc3));
	checkCudaErrors(hipMalloc(&d_Iu, dataSize32f));
	checkCudaErrors(hipMalloc(&d_Iz, dataSize32f));
	// Output Disparity
	checkCudaErrors(hipMalloc(&d_u, dataSize32f));
	checkCudaErrors(hipMalloc(&d_du, dataSize32f));
	checkCudaErrors(hipMalloc(&d_us, dataSize32f));
	// Output Depth
	checkCudaErrors(hipMalloc(&d_depth, dataSize32f));
	// Warping Variables
	checkCudaErrors(hipMalloc(&d_warpUV, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_dwarpUV, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_warpUVs, dataSize32fc2));

	// Vector Fields
	checkCudaErrors(hipMalloc(&d_tvForward, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_tvBackward, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_tv2, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_cv, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_i1calibrated, dataSize32f));

	// Process variables
	checkCudaErrors(hipMalloc(&d_p, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_ps, dataSize32fc2));
	checkCudaErrors(hipMalloc(&d_u_, dataSize32f));
	checkCudaErrors(hipMalloc(&d_u_last, dataSize32f));
	checkCudaErrors(hipMalloc(&d_u_s, dataSize32f));

	// PlaneSweep
	checkCudaErrors(hipMalloc(&ps_i1warp, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_i1warps, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_error, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_depth, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_disparityForward, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_disparityBackward, dataSize32f));
	checkCudaErrors(hipMalloc(&ps_disparityFinal, dataSize32f));

	// 3D
	checkCudaErrors(hipMalloc(&d_X, dataSize32fc3));

	// Debugging
	checkCudaErrors(hipMalloc(&debug_depth, dataSize32f));
	checkCudaErrors(hipMalloc(&d_uvrgb, dataSize32fc3));

	depth = cv::Mat(height, stride, CV_32F);
	warpUV = cv::Mat(height, stride, CV_32FC2);
	warpUVrgb = cv::Mat(height, stride, CV_32FC3);

	return 0;
}

int StereoLite::loadVectorFields(cv::Mat translationVector, cv::Mat calibrationVector) {
	// Padding
	cv::Mat translationVectorPad = cv::Mat(height, stride, CV_32FC2);
	cv::Mat calibrationVectorPad = cv::Mat(height, stride, CV_32FC2);
	cv::copyMakeBorder(translationVector, translationVectorPad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);
	cv::copyMakeBorder(calibrationVector, calibrationVectorPad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);

	// Translation Vector Field
	//translationVector = cv::Mat(height, stride, CV_32FC2);
	//calibrationVector = cv::Mat(height, stride, CV_32FC2);

	checkCudaErrors(hipMemcpy(d_tvForward, (float2 *)translationVectorPad.ptr(), dataSize32fc2, hipMemcpyHostToDevice));

	pTvForward[0] = d_tvForward;
	ScalarMultiply(d_tvForward, -1.0f, width, height, stride, d_tvBackward);
	pTvBackward[0] = d_tvBackward;
	for (int level = 1; level < nLevels; level++) {
		//std::cout << "vectorfields " << pW[level] << " " << pH[level] << " " << pS[level] << std::endl;
		Downscale(pTvForward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvForward[level]);
		Downscale(pTvBackward[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pTvBackward[level]);
	}

	// Calibration Vector Field
	checkCudaErrors(hipMemcpy(d_cv, (float2 *)calibrationVectorPad.ptr(), dataSize32fc2, hipMemcpyHostToDevice));
	return 0;
}

int StereoLite::copyImagesToDevice(cv::Mat i0, cv::Mat i1) {
	// Padding
	cv::copyMakeBorder(i0, im0pad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);
	cv::copyMakeBorder(i1, im1pad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);

	if (i0.type() == CV_8U) {
		checkCudaErrors(hipMemcpy(d_i08u, (uchar *)im0pad.ptr(), dataSize8u, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_i18u, (uchar *)im1pad.ptr(), dataSize8u, hipMemcpyHostToDevice));
		// Convert to 32F
		Cv8uToGray(d_i08u, pI0[0], width, height, stride);
		Cv8uToGray(d_i18u, pI1[0], width, height, stride);
	}
	else if (i0.type() == CV_32F) {
		checkCudaErrors(hipMemcpy(pI0[0], (float *)im0pad.ptr(), dataSize32f, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(pI1[0], (float *)im1pad.ptr(), dataSize32f, hipMemcpyHostToDevice));
	}
	else if (i0.type() == CV_8UC3) {
		checkCudaErrors(hipMemcpy(d_i08uc3, (uchar3 *)im0pad.ptr(), dataSize8uc3, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_i18uc3, (uchar3 *)im1pad.ptr(), dataSize8uc3, hipMemcpyHostToDevice));
		// Convert to 32F
		Cv8uc3ToGray(d_i08uc3, pI0[0], width, height, stride);
		Cv8uc3ToGray(d_i18uc3, pI1[0], width, height, stride);
	}
	return 0;
}

int StereoLite::copyMaskToDevice(cv::Mat mask) {
	cv::copyMakeBorder(mask, fisheyeMaskPad, 0, 0, 0, stride - width, cv::BORDER_CONSTANT, 0);
	checkCudaErrors(hipMemcpy(pFisheyeMask[0], (float *)fisheyeMaskPad.ptr(), dataSize32f, hipMemcpyHostToDevice));

	for (int level = 1; level < nLevels; level++) {
		//std::cout << pW[level] << " " << pH[level] << " " << pS[level] << std::endl;
		DownscaleNearestNeighbor(pFisheyeMask[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pFisheyeMask[level]);
		//DEBUGIMAGE("maskasdfadf", pFisheyeMask[level], pH[level], pS[level], true, true);
	}
	return 0;
}

int StereoLite::solveStereoForwardMasked() {
	// Warp i1 using vector fields=
	WarpImage(pI1[0], width, height, stride, d_cv, d_i1calibrated);
	Swap(pI1[0], d_i1calibrated);

	checkCudaErrors(hipMemset(d_u, 0, dataSize32f));
	checkCudaErrors(hipMemset(d_u_, 0, dataSize32f));
	checkCudaErrors(hipMemset(d_warpUV, 0, dataSize32fc2));

	// Construct pyramid
	for (int level = 1; level < nLevels; level++) {
		Downscale(pI0[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pI0[level]);
		Downscale(pI1[level - 1], pW[level - 1], pH[level - 1], pS[level - 1],
			pW[level], pH[level], pS[level], pI1[level]);
	}

	// Solve stereo
	for (int level = nLevels - 1; level >= 0; level--) {
		if (level == nLevels - 1) {
			ComputeOpticalFlowVector(d_u, pTvForward[level], pW[level], pH[level], pS[level], d_warpUV);
		}

		for (int warpIter = 0; warpIter < nWarpIters; warpIter++) {
			checkCudaErrors(hipMemset(d_p, 0, dataSize32fc2));
			checkCudaErrors(hipMemset(d_du, 0, dataSize32f));

			FindWarpingVector(d_warpUV, pTvForward[level], pW[level], pH[level], pS[level], d_tv2);
			WarpImage(pI1[level], pW[level], pH[level], pS[level], d_warpUV, d_i1warp);

			ComputeDerivativesFisheye(pI0[level], d_i1warp, pTvForward[level],
				pW[level], pH[level], pS[level], d_Iu, d_Iz);

			Clone(d_u_last, pW[level], pH[level], pS[level], d_u_);

			// Inner iteration
			for (int iter = 0; iter < nSolverIters; iter++) {
				// Solve Problem1A
				ThresholdingL1Masked(pFisheyeMask[level], d_u, d_u_, d_Iu, d_Iz, lambda, theta, d_us, pW[level], pH[level], pS[level]);
				Swap(d_u, d_us);

				// Solve Problem1B
				SolveProblem1bMasked(pFisheyeMask[level], d_u, d_p, theta, d_u_, pW[level], pH[level], pS[level]);

				// Solve Problem2
				SolveProblem2Masked(pFisheyeMask[level], d_u_, d_p, theta, tau, d_ps, pW[level], pH[level], pS[level]);
				Swap(d_p, d_ps);
			}
			Subtract(d_u_, d_u_last, pW[level], pH[level], pS[level], d_du);
			LimitRange(d_du, limitRange, pW[level], pH[level], pS[level], d_du);
			Add(d_u_last, d_du, pW[level], pH[level], pS[level], d_u_);
			Clone(d_u, pW[level], pH[level], pS[level], d_u_);

			ComputeOpticalFlowVector(d_du, d_tv2, pW[level], pH[level], pS[level], d_dwarpUV);
			Add(d_warpUV, d_dwarpUV, pW[level], pH[level], pS[level], d_warpUV);
		}

		// Upscale
		if (level > 0)
		{
			float scale = fScale;
			Upscale(d_u, pW[level], pH[level], pS[level],
				pW[level - 1], pH[level - 1], pS[level - 1], scale, d_us);
			Upscale(d_u_, pW[level], pH[level], pS[level],
				pW[level - 1], pH[level - 1], pS[level - 1], scale, d_u_s);
			Upscale(d_warpUV, pW[level], pH[level], pS[level],
				pW[level - 1], pH[level - 1], pS[level - 1], scale, d_warpUVs);

			Swap(d_u, d_us);
			Swap(d_u_, d_u_s);
			Swap(d_warpUV, d_warpUVs);
		}
	}

	return 0;
}

int StereoLite::copyStereoToHost(cv::Mat &wCropped) {
	// Convert Disparity to Depth
	ConvertDisparityToDepth(d_u, baseline, focal, width, height, stride, d_depth);

	// Remove Padding
	//checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_w, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_depth, dataSize32f, hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	wCropped = depth(roi);
	return 0;
}


// PlaneSweep
int StereoLite::planeSweepForward() {
	// Plane sweep on level=1
	int planeSweepLevel = 0;
	checkCudaErrors(hipMemset(ps_error, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_disparityForward, 0, dataSize32f));
	Clone(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], pI1[planeSweepLevel]);
	SetValue(ps_error, planeSweepMaxError, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI0[planeSweepLevel], ps_disparityForward, sweep, planeSweepWindow,
			pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], d_tvForward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}
	return 0;
}

int StereoLite::planeSweepBackward() {
	// Plane sweep on level=1
	int planeSweepLevel = 0;
	checkCudaErrors(hipMemset(ps_error, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_disparityBackward, 0, dataSize32f));
	Clone(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], pI0[planeSweepLevel]);
	SetValue(ps_error, planeSweepMaxError, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI1[planeSweepLevel], ps_disparityBackward, sweep, planeSweepWindow,
			pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[planeSweepLevel], pH[planeSweepLevel], pS[planeSweepLevel], d_tvBackward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}
	return 0;
}

int StereoLite::planeSweep() {
	// Forward
	int lvl = 0;
	checkCudaErrors(hipMemset(ps_error, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_disparityForward, 0, dataSize32f));
	Clone(ps_i1warp, pW[lvl], pH[lvl], pS[lvl], pI1[lvl]);
	SetValue(ps_error, planeSweepMaxError, pW[lvl], pH[lvl], pS[lvl]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI0[lvl], ps_disparityForward, sweep, planeSweepWindow,
			pW[lvl], pH[lvl], pS[lvl], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[lvl], pH[lvl], pS[lvl], d_tvForward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}

	// Backward
	checkCudaErrors(hipMemset(ps_error, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_depth, 0, dataSize32f));
	checkCudaErrors(hipMemset(ps_disparityBackward, 0, dataSize32f));
	Clone(ps_i1warp, pW[lvl], pH[lvl], pS[lvl], pI0[lvl]);
	SetValue(ps_error, planeSweepMaxError, pW[lvl], pH[lvl], pS[lvl]);
	for (int sweep = 0; sweep < planeSweepMaxDisparity; sweep += planeSweepStride) {
		PlaneSweepCorrelation(ps_i1warp, pI1[lvl], ps_disparityBackward, sweep, planeSweepWindow,
			pW[lvl], pH[lvl], pS[lvl], ps_error);
		for (int psStride = 0; psStride < planeSweepStride; psStride++) {
			WarpImage(ps_i1warp, pW[lvl], pH[lvl], pS[lvl], d_tvBackward, ps_i1warps);
			Swap(ps_i1warp, ps_i1warps);
		}
	}

	// Left-Right Consistency
	

	return 0;
}

int StereoLite::copyPlanesweepForwardToHost(cv::Mat &wCropped) {
	// Convert Disparity to Depth
	ConvertDisparityToDepth(ps_disparityForward, baseline, focal, width, height, stride, d_depth);

	// Remove Padding
	//checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_w, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_depth, dataSize32f, hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	wCropped = depth(roi);
	return 0;
}

int StereoLite::copyPlanesweepBackwardToHost(cv::Mat &wCropped) {
	// Convert Disparity to Depth
	ConvertDisparityToDepth(ps_disparityBackward, baseline, focal, width, height, stride, d_depth);

	// Remove Padding
	//checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_w, stride * height * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy((float *)depth.ptr(), d_depth, dataSize32f, hipMemcpyDeviceToHost));
	cv::Rect roi(0, 0, width, height); // define roi here as x0, y0, width, height
	wCropped = depth(roi);
	return 0;
}

// Utilities
int StereoLite::iAlignUp(int n)
{
	int m = this->StrideAlignment;
	int mod = n % m;

	if (mod)
		return n + m - mod;
	else
		return n;
}

int StereoLite::iDivUp(int n, int m)
{
	return (n + m - 1) / m;
}

template<typename T> void StereoLite::Swap(T &a, T &ax)
{
	T t = a;
	a = ax;
	ax = t;
}

template<typename T> void StereoLite::Copy(T &dst, T &src)
{
	dst = src;
}