#include "hip/hip_runtime.h"
#include "flow.h"

//*****************
// Warping
//*****************
texture<float, hipTextureType2D, hipReadModeElementType> FlowTexToWarp;

__global__ void FlowWarpingKernel(int width, int height, int stride,
	const float2 *warpUV, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + warpUV[pos].x + 0.5f) / (float)width;
	float y = ((float)iy + warpUV[pos].y + 0.5f) / (float)height;

	out[pos] = tex2D(FlowTexToWarp, x, y);
}

void Flow::WarpImage(const float *src, int w, int h, int s,
	const float2 *warpUV, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	FlowTexToWarp.addressMode[0] = hipAddressModeMirror;
	FlowTexToWarp.addressMode[1] = hipAddressModeMirror;
	FlowTexToWarp.filterMode = hipFilterModeLinear;
	FlowTexToWarp.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, FlowTexToWarp, src, w, h, s * sizeof(float));

	FlowWarpingKernel << <blocks, threads >> > (w, h, s, warpUV, out);
}


//*****************
// Image Converter
//*****************
__global__
void FlowCv8uToGrayKernel(uchar *d_iCv8u, float *d_iGray, int width, int height, int stride)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int c = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((r < height) && (c < width))
	{
		int idx = c + stride * r;        // current pixel index 

		//d_iGray[idx] = 0.2126f * (float)pixel.x + 0.7152f * (float)pixel.y + 0.0722f * (float)pixel.z;
		d_iGray[idx] = (float)d_iCv8u[idx] / 256.0f;
	}
}

void Flow::Cv8uToGray(uchar * d_iCv8u, float *d_iGray, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowCv8uToGrayKernel << < blocks, threads >> > (d_iCv8u, d_iGray, w, h, s);
}

__global__
void FlowCv8uc3ToGrayKernel(uchar3 *d_iRgb, float *d_iGray, int width, int height, int stride)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int c = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((r < height) && (c < width))
	{
		int idx = c + stride * r;        // current pixel index 

		uchar3 pixel = d_iRgb[idx];

		//d_iGray[idx] = 0.2126f * (float)pixel.x + 0.7152f * (float)pixel.y + 0.0722f * (float)pixel.z;
		d_iGray[idx] = ((float)pixel.x + (float)pixel.y + (float)pixel.z) / 3;
		d_iGray[idx] = d_iGray[idx] / 256.0f;
	}
}

void Flow::Cv8uc3ToGray(uchar3 * d_iRgb, float *d_iGray, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowCv8uc3ToGrayKernel << < blocks, threads >> > (d_iRgb, d_iGray, w, h, s);
}


//**********************
// TVL1 Direct Alignment
//**********************
// Solve Problem1A - Thresholding
__global__ void FlowThresholdingL1MaskedKernel(float* mask, float2* u, float2* u_med, float* Ix, float* Iy, float* Iz,
	float lambda, float theta, int width, int height, int stride)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	// Thresholding
	float du_med = (u[pos].x - u_med[pos].x);
	float dv_med = (u[pos].y - u_med[pos].y);

	float Ixs = Ix[pos];
	float Iys = Iy[pos];
	float Izs = Iz[pos];

	float divisor = Ixs * Ixs + Iys * Iys;
	float rho = Ixs * du_med + Iys * dv_med + Izs;
	float upper = lambda * theta * divisor;
	float lower = -lambda * theta * divisor;

	float du;
	float dv;

	if ((rho <= upper) && (rho >= lower)) {
		if (divisor == 0) {
			du = du_med;
			dv = dv_med;
		}
		else {
			du = du_med - rho * Ixs / divisor;
			dv = dv_med - rho * Iys / divisor;
		}
	}
	else if (rho < lower) {
		du = du_med + lambda * theta * Ixs;
		dv = dv_med + lambda * theta * Iys;
	}
	else if (rho > upper) {
		du = du_med - lambda * theta *Ixs;
		dv = dv_med - lambda * theta *Iys;
	}

	u[pos].x = u_med[pos].x + du;
	u[pos].y = u_med[pos].y + dv;
}

void Flow::ThresholdingL1Masked(float* mask, float2* u, float2* u_med, float* Ix, float* Iy, float* Iz,
	float lambda, float theta, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowThresholdingL1MaskedKernel << < blocks, threads >> > (mask, u, u_med, Ix, Iy, Iz, lambda, theta,
		w, h, s);
}

__global__ void FlowSparsePriorL1Kernel(float* mask, float* u, float* u_, float* usparse, float* Iu, float* Iz, float lambda, float l2lambda,
	float* lambdaMask, float theta,
	int width, int height, int stride, float* us)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	// Thresholding
	float u_pos = u_[pos];
	float dun = (u[pos] - u_pos);
	float Ius = Iu[pos];
	float rho = Ius * dun + Iz[pos];

	float upper = lambda * theta * (Ius * Ius);
	float lower = -lambda * theta *(Ius * Ius);
	float du;

	if ((rho <= upper) && (rho >= lower)) {
		if (Ius == 0) {
			du = dun;
		}
		else {
			du = dun - rho / Ius;
		}
	}
	else if (rho < lower) {
		du = dun + lambda * theta *Ius;
	}
	else if (rho > upper) {
		du = dun - lambda * theta *Ius;
	}

	float ul1 = u_pos + du;
	float usparsepos = usparse[pos];
	/*if (abs(ul1 - usparsepos) > 5.0f) {
		usparsepos = ul1;
	}*/

	if (lambdaMask[pos] == 0.0f) {
		us[pos] = ul1;
	}
	else {
		us[pos] = (ul1 + l2lambda * usparsepos) / (1.0f + l2lambda);
	}
}

void Flow::SparsePriorL1(float* fisheyeMask, float* u, float* u_, float * usparse, float* Iu, float* Iz,
	float lambda, float l2lambda, float* lambdaMask, float theta,
	float* us, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowSparsePriorL1Kernel << < blocks, threads >> > (fisheyeMask, u, u_, usparse, Iu, Iz, lambda, l2lambda,
		lambdaMask, theta,
		w, h, s, us);
}


__global__ void FlowL2Kernel(float* mask, float* u, float* u_, float l2lambda, float* lambdaMask,
	int width, int height, int stride, float* us)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	if (lambdaMask[pos] == 0.0f) {
		us[pos] = u_[pos];
	}
	else {
		us[pos] = (u_[pos] + l2lambda * u[pos]) / (1.0f + l2lambda);
	}
}

void Flow::SimpleL2(float* mask, float* u, float* u_, float l2lambda, float* lambdaMask,
	float* us, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowL2Kernel << < blocks, threads >> > (mask, u, u_, l2lambda, lambdaMask, w, h, s, us);
}

// Solve Problem1B
__global__ void FlowSolveProblem1bMaskedKernel(float* mask, float2* u, float2 *pu, float2* pv, float theta,
	int width, int height, int stride, float2* umed)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	int left = (ix - 1) + iy * stride;
	int up = ix + (iy - 1) * stride;

	float maskLeft, maskUp;
	if (ix - 1 < 0) {
		maskLeft = 0.0f;
	}
	else maskLeft = mask[left];

	if (iy - 1 < 0) {
		maskUp = 0.0f;
	}
	else maskUp = mask[up];

	float divpu, divpv;
	float2 pupos = pu[pos];
	float2 pvpos = pv[pos];
	if (maskLeft == 0.0f) {
		if (maskUp == 0.0f) {
			//divp = ppos.x + ppos.y;
			divpu = 0.0f;
			divpv = 0.0f;
		}
		else {
			//divp = ppos.x + ppos.y - p[up].y;
			divpu = pupos.y - pu[up].y;
			divpv = pvpos.y - pv[up].y;
		}
	}
	else {
		if (maskUp == 0.0f) {
			//divp = ppos.x - p[left].x + ppos.y;
			divpu = pupos.x - pu[left].x;
			divpv = pvpos.x - pv[left].x;
		}
		else {
			divpu = pupos.x - pu[left].x + pupos.y - pu[up].y;
			divpv = pvpos.x - pv[left].x + pvpos.y - pv[up].y;
		}
	}

	umed[pos].x = u[pos].x + theta * divpu;
	umed[pos].y = u[pos].y + theta * divpv;
}

void Flow::SolveProblem1bMasked(float* mask, float2* u, float2 *pu, float2* pv, float theta,
	float2* umed, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowSolveProblem1bMaskedKernel << < blocks, threads >> > (mask, u, pu, pv, theta,
		w, h, s, umed);
}

// Solve Problem 2
__global__ void FlowSolveProblem2MaskedKernel(float* mask, float2* u, float2 *pu, float2* pv, float theta, float tau,
	int width, int height, int stride, float2* pus, float2* pvs)
{
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy >= height) && (ix >= width)) return;
	int pos = ix + iy * stride;
	if (mask[pos] == 0.0f) return;

	int right = (ix + 1) + iy * stride;
	int down = ix + (iy + 1) * stride;
	int left = (ix - 1) + iy * stride;
	int up = ix + (iy - 1) * stride;
	float maskRight, maskDown;
	if (ix + 1 >= width) {
		maskRight = 0.0f;
	}
	else maskRight = mask[right];
	if (iy + 1 >= height) {
		maskDown = 0.0f;
	}
	else maskDown = mask[down];

	float dux, duy;
	float dvx, dvy;
	if (maskRight == 0.0f) {
		dux = 0;
		dvx = 0;
	}
	else {
		dux = u[right].x - u[pos].x;
		dvx = u[right].y - u[pos].y;
	}
	if (maskDown == 0.0f) {
		duy = 0;
		dvy = 0;
	}
	else {
		duy = u[down].x - u[pos].x;
		dvy = u[down].y - u[pos].y;
	}

	float magdu = sqrt(dux * dux + duy * duy);
	float magdv = sqrt(dvx * dvx + dvy * dvy);
	float fac = tau / theta;

	float2 pusub = pu[pos];
	float2 pvsub = pv[pos];

	pus[pos].x = (pusub.x + fac * dux) / (1 + fac * magdu);
	pus[pos].y = (pusub.y + fac * duy) / (1 + fac * magdu);
	pvs[pos].x = (pvsub.x + fac * dvx) / (1 + fac * magdv);
	pvs[pos].y = (pvsub.y + fac * dvy) / (1 + fac * magdv);
}

void Flow::SolveProblem2Masked(float* mask, float2* u, float2 *pu, float2 *pv, float theta, float tau,
	float2* pus, float2* pvs, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	FlowSolveProblem2MaskedKernel << < blocks, threads >> > (mask, u, pu, pv, theta, tau, w, h, s, pus, pvs);
}


//********************
// COMPUTE DERIVATIVES
//********************

texture<float, hipTextureType2D, hipReadModeElementType> texI0;
texture<float, hipTextureType2D, hipReadModeElementType> texI1;

__global__ void DirectComputeDerivativesKernel(int width, int height, int stride,
	float *Ix, float *Iy, float *Iz)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	float t0, t1;
	// x derivative
	t0 = tex2D(texI0, x - 2.0f * dx, y);
	t0 -= tex2D(texI0, x - 1.0f * dx, y) * 8.0f;
	t0 += tex2D(texI0, x + 1.0f * dx, y) * 8.0f;
	t0 -= tex2D(texI0, x + 2.0f * dx, y);
	t0 /= 12.0f;

	t1 = tex2D(texI1, x - 2.0f * dx, y);
	t1 -= tex2D(texI1, x - 1.0f * dx, y) * 8.0f;
	t1 += tex2D(texI1, x + 1.0f * dx, y) * 8.0f;
	t1 -= tex2D(texI1, x + 2.0f * dx, y);
	t1 /= 12.0f;

	Ix[pos] = (t0 + t1) * 0.5f;

	// t derivative
	Iz[pos] = tex2D(texI1, x, y) - tex2D(texI0, x, y);

	// y derivative
	t0 = tex2D(texI0, x, y - 2.0f * dy);
	t0 -= tex2D(texI0, x, y - 1.0f * dy) * 8.0f;
	t0 += tex2D(texI0, x, y + 1.0f * dy) * 8.0f;
	t0 -= tex2D(texI0, x, y + 2.0f * dy);
	t0 /= 12.0f;

	t1 = tex2D(texI1, x, y - 2.0f * dy);
	t1 -= tex2D(texI1, x, y - 1.0f * dy) * 8.0f;
	t1 += tex2D(texI1, x, y + 1.0f * dy) * 8.0f;
	t1 -= tex2D(texI1, x, y + 2.0f * dy);
	t1 /= 12.0f;

	Iy[pos] = (t0 + t1) * 0.5f;
}

///CUDA CALL FUNCTIONS ***********************************************************
void Flow::ComputeDerivatives(float *I0, float *I1,
	int w, int h, int s,
	float *Ix, float *Iy, float *Iz)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	// mirror if a coordinate value is out-of-range
	texI0.addressMode[0] = hipAddressModeMirror;
	texI0.addressMode[1] = hipAddressModeMirror;
	texI0.filterMode = hipFilterModeLinear;
	texI0.normalized = true;

	texI1.addressMode[0] = hipAddressModeMirror;
	texI1.addressMode[1] = hipAddressModeMirror;
	texI1.filterMode = hipFilterModeLinear;
	texI1.normalized = true;

	//hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(0, texI0, I0, w, h, s * sizeof(float));
	hipBindTexture2D(0, texI1, I1, w, h, s * sizeof(float));

	DirectComputeDerivativesKernel << < blocks, threads >> > (w, h, s, Ix, Iy, Iz);
}

//****************
// Gradient
//****************

texture<float, 2, hipReadModeElementType> texForGradient;

__global__ void FlowGradientKernel(float* output, int width, int height, int stride) {
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;

		float dx = 1.0f / (float)width;
		float dy = 1.0f / (float)height;

		float x = ((float)ix + 0.5f) * dx;
		float y = ((float)iy + 0.5f) * dy;

		float2 grad;
		float t0;
		// x derivative
		t0 = tex2D(texForGradient, x + 1.0f * dx, y);
		t0 -= tex2D(texForGradient, x, y);
		t0 = tex2D(texForGradient, x + 1.0f * dx, y + 1.0f * dy);
		t0 -= tex2D(texForGradient, x, y + 1.0f * dy);
		grad.x = t0;

		// y derivative
		t0 = tex2D(texForGradient, x, y + 1.0f * dy);
		t0 -= tex2D(texForGradient, x, y);
		t0 = tex2D(texForGradient, x + 1.0f * dx, y + 1.0f * dy);
		t0 -= tex2D(texForGradient, x + 1.0f * dx, y);
		grad.y = t0;

		output[pos] = sqrtf(grad.x * grad.x + grad.y * grad.y);
	}
}


void Flow::Gradient(float* input, int w, int h, int s, float* output) {
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(width, threads.x), iDivUp(height, threads.y));

	// mirror if a coordinate value is out-of-range
	texForGradient.addressMode[0] = hipAddressModeMirror;
	texForGradient.addressMode[1] = hipAddressModeMirror;
	texForGradient.filterMode = hipFilterModeLinear;
	texForGradient.normalized = true;
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D(0, texForGradient, input, w, h, s * sizeof(float));

	FlowGradientKernel << < blocks, threads >> > (output, w, h, s);
}

__global__ void FlowFilterGradientKernel(float* gradient, float2* u, float2* umed, float threshold,
	int width, int height, int stride) {
	int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((iy < height) && (ix < width))
	{
		int pos = ix + iy * stride;
		if (gradient[pos] <= threshold) {
			umed[pos].x = 0.0f;
			umed[pos].y = 0.0f;
			u[pos].x = 0.0f;
			u[pos].y = 0.0f;
		}
	}
}

void Flow::FilterGradient(float* gradient, float2* u, float2* umed, float threshold, int w, int h, int s)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	//flowToHSVKernel << < blocks, threads >> >(u, v, w, h, s, uRGB, flowscale);
	FlowFilterGradientKernel << < blocks, threads >> > (gradient, u, umed, threshold, w, h, s);
}


__global__
void FlowComputeColorKernel(float2 *uv, int width, int height, int stride, float3 *uvRGB, float flowscale) {
	int r = blockIdx.y * blockDim.y + threadIdx.y;        // current row 
	int c = blockIdx.x * blockDim.x + threadIdx.x;        // current column 

	if ((r < height) && (c < width))
	{
		int pos = c + stride * r;
		float du = uv[pos].x / flowscale;
		float dv = uv[pos].y / flowscale;

		int ncols = 55;
		float rad = sqrtf(du * du + dv * dv);
		float a = atan2(-dv, -du) / 3.14159f;
		float fk = (a + 1) / 2 * ((float)ncols - 1);
		int k0 = floorf(fk); //colorwheel index lower bound
		int k1 = k0 + 1; //colorwheel index upper bound
		if (k1 == ncols) {
			k1 = 1;
		}
		float f = fk - (float)k0;

		float colorwheelR[55] = { 255, 255,	255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
			255, 213, 170, 128, 85, 43, 0, 0, 0, 0, 0, 0, 0, 0, 0,
			0, 0, 0, 0, 0, 0, 0, 19, 39, 58, 78, 98, 117, 137, 156,
			176, 196, 215, 235, 255, 255, 255, 255, 255, 255 };
		float colorwheelG[55] = { 0, 17, 34, 51, 68, 85, 102, 119, 136, 153, 170, 187, 204, 221, 238,
			255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 232, 209, 186, 163,
			140, 116, 93, 70, 47, 24, 0, 0, 0, 0, 0, 0, 0, 0, 0,
			0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
		float colorwheelB[55] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
			0, 0, 0, 0, 0, 0, 0, 63, 127, 191, 255, 255, 255, 255, 255,
			255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
			255, 255, 255, 255, 255, 213, 170, 128, 85, 43 };

		float colR = (1 - f) * (colorwheelR[k0] / 255.0f) + f * (colorwheelR[k1] / 255.0f);
		float colG = (1 - f) * (colorwheelG[k0] / 255.0f) + f * (colorwheelG[k1] / 255.0f);
		float colB = (1 - f) * (colorwheelB[k0] / 255.0f) + f * (colorwheelB[k1] / 255.0f);

		if (rad <= 1) {
			colR = 1 - rad * (1 - colR);
			colG = 1 - rad * (1 - colG);
			colB = 1 - rad * (1 - colB);
		}
		else {
			colR = colR * 0.75;
			colG = colG * 0.75;
			colB = colB * 0.75;
		}

		uvRGB[pos].z = (colR);
		uvRGB[pos].y = (colG);
		uvRGB[pos].x = (colB);
	}
}

void Flow::FlowToHSV(float2* uv, int w, int h, int s, float3 * uRGB, float flowscale)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	//flowToHSVKernel << < blocks, threads >> >(u, v, w, h, s, uRGB, flowscale);
	FlowComputeColorKernel << < blocks, threads >> > (uv, w, h, s, uRGB, flowscale);
}


/// image to downscale
texture<float, hipTextureType2D, hipReadModeElementType> texFine;
texture<float2, hipTextureType2D, hipReadModeElementType> texFineFloat2;

// *********************************
// Downscaling
// *********************************
__global__ void FlowDownscaleKernel(int width, int height, int stride, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}

void Flow::Downscale(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModeLinear;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	FlowDownscaleKernel << < blocks, threads >> > (newWidth, newHeight, newStride, out);
}


__global__ void FlowDownscaleNearestNeighborKernel(int width, int height, int stride, float *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height)
	{
		return;
	}

	float dx = 1.0f / (float)width;
	float dy = 1.0f / (float)height;

	float x = ((float)ix + 0.5f) * dx;
	float y = ((float)iy + 0.5f) * dy;

	int pos = ix + iy * stride;

	out[pos] = tex2D(texFine, x, y);
	/*out[pos] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
		tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));*/
}

void Flow::DownscaleNearestNeighbor(const float *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texFine.addressMode[0] = hipAddressModeMirror;
	texFine.addressMode[1] = hipAddressModeMirror;
	texFine.filterMode = hipFilterModePoint;
	texFine.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	checkCudaErrors(hipBindTexture2D(0, texFine, src, width, height, stride * sizeof(float)));

	FlowDownscaleNearestNeighborKernel << < blocks, threads >> > (newWidth, newHeight, newStride, out);
}

//*****************
// Upscaling
//*****************
texture<float2, hipTextureType2D, hipReadModeElementType> texCoarseFloat2;

__global__ void FlowUpscaleFloat2Kernel(int width, int height, int stride, float scale, float2 *out)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	if (ix >= width || iy >= height) return;

	float x = ((float)ix + 0.5f) / (float)width;
	float y = ((float)iy + 0.5f) / (float)height;

	// exploit hardware interpolation
	// and scale interpolated vector to match next pyramid level resolution
	float2 src = tex2D(texCoarseFloat2, x, y);
	out[ix + iy * stride].x = src.x * scale;
	out[ix + iy * stride].y = src.y * scale;
}

void Flow::Upscale(const float2 *src, int width, int height, int stride,
	int newWidth, int newHeight, int newStride, float scale, float2 *out)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(newWidth, threads.x), iDivUp(newHeight, threads.y));

	// mirror if a coordinate value is out-of-range
	texCoarseFloat2.addressMode[0] = hipAddressModeMirror;
	texCoarseFloat2.addressMode[1] = hipAddressModeMirror;
	texCoarseFloat2.filterMode = hipFilterModeLinear;
	texCoarseFloat2.normalized = true;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

	hipBindTexture2D(0, texCoarseFloat2, src, width, height, stride * sizeof(float2));

	FlowUpscaleFloat2Kernel << < blocks, threads >> > (newWidth, newHeight, newStride, scale, out);
}


//************************
// Median Filter
//************************
__global__
void FlowMedianFilterKernel5(float2* u, int width, int height, int stride,
	float2 *outputu)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float mu[25] = { 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0 };

	float mv[25] = { 0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0,
		0, 0, 0, 0, 0 };

	for (int j = 0; j < 5; j++) {
		for (int i = 0; i < 5; i++) {
			//get values
			int col = (ix + i - 2);
			int row = (iy + j - 2);
			if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
				mu[j * 5 + i] = u[col + stride * row].x;
				mv[j * 5 + i] = u[col + stride * row].y;
			}
			else if ((col < 0) && (row >= 0) && (row < height)) {
				mu[j * 5 + i] = u[stride*row].x;
				mv[j * 5 + i] = u[stride*row].y;
			}
			else if ((col >= width) && (row >= 0) && (row < height)) {
				mu[j * 5 + i] = u[width - 1 + stride * row].x;
				mv[j * 5 + i] = u[width - 1 + stride * row].y;
			}
			else if ((col >= 0) && (col < width) && (row < 0)) {
				mu[j * 5 + i] = u[col].x;
				mv[j * 5 + i] = u[col].y;
			}
			else if ((col >= 0) && (col < width) && (row >= height)) {
				mu[j * 5 + i] = u[col + stride * (height - 1)].x;
				mv[j * 5 + i] = u[col + stride * (height - 1)].y;
			}
			//solve gaussian
		}
	}

	float tmpu, tmpv;
	for (int j = 0; j < 13; j++) {
		for (int i = j + 1; i < 25; i++) {
			if (mu[j] > mu[i]) {
				//Swap the variables.
				tmpu = mu[j];
				mu[j] = mu[i];
				mu[i] = tmpu;
			}
			if (mv[j] > mv[i]) {
				//Swap the variables.
				tmpv = mv[j];
				mv[j] = mv[i];
				mv[i] = tmpv;
			}
		}
	}

	outputu[pos].x = mu[12];
	outputu[pos].y = mv[12];
}

__global__ void FlowMedianFilterKernel3(float2* u,
	int width, int height, int stride,
	float2 *outputu)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	if (ix >= width || iy >= height) return;

	float mu[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	float mv[9] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	for (int j = 0; j < 3; j++) {
		for (int i = 0; i < 3; i++) {
			//get values
			int col = (ix + i - 1);
			int row = (iy + j - 1);
			int index = j * 3 + i;
			if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
				mu[index] = u[col + stride * row].x;
				mv[index] = u[col + stride * row].y;
			}
			else if ((col < 0) && (row >= 0) && (row < height)) {
				mu[index] = u[stride*row].x;
				mv[index] = u[stride*row].y;
			}
			else if ((col > width) && (row >= 0) && (row < height)) {
				mu[index] = u[width - 1 + stride * row].x;
				mv[index] = u[width - 1 + stride * row].y;
			}
			else if ((col >= 0) && (col < width) && (row < 0)) {
				mu[index] = u[col].x;
				mv[index] = u[col].y;
			}
			else if ((col >= 0) && (col < width) && (row > height)) {
				mu[index] = u[col + stride * (height - 1)].x;
				mv[index] = u[col + stride * (height - 1)].y;
			}
			//solve gaussian
		}
	}

	float tmpu, tmpv;
	for (int j = 0; j < 9; j++) {
		for (int i = j + 1; i < 9; i++) {
			if (mu[j] > mu[i]) {
				//Swap the variables.
				tmpu = mu[j];
				mu[j] = mu[i];
				mu[i] = tmpu;
			}
			if (mv[j] > mv[i]) {
				//Swap the variables.
				tmpv = mv[j];
				mv[j] = mv[i];
				mv[i] = tmpv;
			}
		}
	}

	outputu[pos].x= mu[4];
	outputu[pos].y = mv[4];
}


void Flow::MedianFilter(float2 *inputu,	int w, int h, int s, float2 *outputu, int kernelsize)
{
	dim3 threads(BlockWidth, BlockHeight);
	dim3 blocks(iDivUp(w, threads.x), iDivUp(h, threads.y));

	if (kernelsize == 3) {
		FlowMedianFilterKernel3 << < blocks, threads >> > (inputu, w, h, s, outputu);
	}
	else if (kernelsize == 5) {
		FlowMedianFilterKernel5 << < blocks, threads >> > (inputu, w, h, s, outputu);
	}
}